
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include <iostream>
__global__
void add(float *x, float *y, int n ){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride){
      y[i] = x[i] + y[i];
    }

}

int main(){
    int N = 1 << 20;
    float *a, *b;
    hipMallocManaged(&a, N * sizeof(int));
    hipMallocManaged(&b, N * sizeof(int));
  
    for (int i = 0; i < N; i++){
        a[i] = i * 3.0;
        b[i] = i * 5.0;
    }        
    clock_t start = clock();
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
 
    add<<<numBlocks, blockSize>>>( a, b, N);
    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++){
        maxError = fmax(maxError, fabs(b[i]-((i*3.0f )+ (i*5.0f))));
    }
      std::cout << "Max error: " << maxError << std::endl;
    

    clock_t end = clock();
  

    double time = (double)(end - start)/ CLOCKS_PER_SEC;
    std::cout << "Time taken: " << time << std::endl;
  

    delete[] a;
    delete[] b;
    return 0;



}